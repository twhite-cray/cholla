
#include <hip/hip_runtime.h>
/*! \file CTU_2D_cuda.cu
 *  \brief Definitions of the cuda 2D CTU algorithm functions. */

#ifdef CUDA

#include<stdio.h>
#include<math.h>
#include<cuda.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"CTU_2D_cuda.h"
#include"pcm_cuda.h"
#include"plmp_cuda.h"
#include"plmc_cuda.h"
#include"ppmp_cuda.h"
#include"ppmc_cuda.h"
#include"exact_cuda.h"
#include"roe_cuda.h"
#include"hllc_cuda.h"
#include"h_correction_2D_cuda.h"
#include"cooling_cuda.h"
#include"subgrid_routines_2D.h"



__global__ void Evolve_Interface_States_2D(Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F1_x,
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F1_y,
                                           int nx, int ny, int n_ghost, Real dx, Real dy, Real dt, int n_fields);


Real CTU_Algorithm_2D_CUDA(Real *host_conserved0, Real *host_conserved1, int nx, int ny, int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound, Real ybound, Real dt, int n_fields)
{

  //Here, *host_conserved contains the entire
  //set of conserved variables on the grid
  //concatenated into a 1-d array
  //host_conserved0 contains the values at time n,
  //host_conserved1 will contain the values at time n+1

  // Initialize dt values
  Real max_dti = 0;
  #ifdef COOLING_GPU
  Real min_dt = 1e10;
  #endif

  if ( !block_size ) {
    // calculate the dimensions for each subgrid block
    sub_dimensions_2D(nx, ny, n_ghost, &nx_s, &ny_s, &block1_tot, &block2_tot, &remainder1, &remainder2, n_fields);
    //printf("%d %d %d %d %d %d\n", nx_s, ny_s, block1_tot, block2_tot, remainder1, remainder2);
    nz_s = 1;
    block_tot = block1_tot*block2_tot;
    // number of cells in one subgrid block
    BLOCK_VOL = nx_s*ny_s*nz_s;
    // dimensions for the 1D GPU grid
    ngrid = (BLOCK_VOL + TPB - 1) / (TPB);
    #ifndef DYNAMIC_GPU_ALLOC
    block_size = true;
    #endif
  }
  // set values for GPU kernels
  // number of blocks per 1D grid  
  dim3 dim2dGrid(ngrid, 1, 1);
  //number of threads per 1D block   
  dim3 dim1dBlock(TPB, 1, 1);

  // Set up pointers for the location to copy from and to
  if (block_tot == 1) {
    tmp1 = host_conserved0;
    tmp2 = host_conserved1;
  }  

  if ( !memory_allocated ) {

    // allocate buffer to copy conserved variable blocks from and to 
    if (block_tot > 1) {
      if ( NULL == ( buffer = (Real *) malloc(n_fields*BLOCK_VOL*sizeof(Real)) ) ) {
        printf("Failed to allocate CPU buffer.\n");
      }
      tmp1 = buffer;
      tmp2 = buffer;
    }
    // allocate an array on the CPU to hold max_dti returned from each thread block
    host_dti_array = (Real *) malloc(ngrid*sizeof(Real));
    #ifdef COOLING_GPU
    host_dt_array = (Real *) malloc(ngrid*sizeof(Real));
    #endif  
  
    // allocate memory on the GPU
    CudaSafeCall( cudaMalloc((void**)&dev_conserved, n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Lx, n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Rx, n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Ly, n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&Q_Ry, n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_x,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&F_y,  n_fields*BLOCK_VOL*sizeof(Real)) );
    CudaSafeCall( cudaMalloc((void**)&dev_dti_array, ngrid*sizeof(Real)) );
    #ifdef COOLING_GPU
    CudaSafeCall( cudaMalloc((void**)&dev_dt_array, ngrid*sizeof(Real)) );
    #endif 

    #ifndef DYNAMIC_GPU_ALLOC 
    // If memory is single allocated: memory_allocated becomes true and succesive timesteps won't allocate memory.
    // If the memory is not single allocated: memory_allocated remains Null and memory is allocated every timestep.
    memory_allocated = true;
    #endif 
  }

  // counter for which block we're on
  int block = 0;


  // START LOOP OVER SUBGRID BLOCKS HERE
  while (block < block_tot) {

    // copy the conserved variable block to the buffer
    host_copy_block_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved0, buffer, n_fields);

    // calculate the global x and y offsets of this subgrid block
    // (only needed for gravitational potential)
    get_offsets_2D(nx_s, ny_s, n_ghost, x_off, y_off, block, block1_tot, block2_tot, remainder1, remainder2, &x_off_s, &y_off_s);    

    // copy the conserved variables onto the GPU
    CudaSafeCall( cudaMemcpy(dev_conserved, tmp1, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyHostToDevice) );


    // Step 1: Do the reconstruction
    #ifdef PCM
    PCM_Reconstruction_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, nx_s, ny_s, n_ghost, gama, n_fields);
    #endif
    #ifdef PLMP
    PLMP_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PLMP_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    #endif
    #ifdef PLMC
    PLMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PLMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    #endif
    #ifdef PPMP
    PPMP_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PPMP_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    #endif
    #ifdef PPMC
    PPMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Lx, Q_Rx, nx_s, ny_s, nz_s, n_ghost, dx, dt, gama, 0, n_fields);
    PPMC_cuda<<<dim2dGrid,dim1dBlock>>>(dev_conserved, Q_Ly, Q_Ry, nx_s, ny_s, nz_s, n_ghost, dy, dt, gama, 1, n_fields);
    #endif
    CudaCheckError();


    // Step 2: Calculate the fluxes
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    #endif
    #ifdef HLLC 
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    #endif
    CudaCheckError();

#ifdef CTU

    // Step 3: Evolve the interface states
    Evolve_Interface_States_2D<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, Q_Ly, Q_Ry, F_y, nx_s, ny_s, n_ghost, dx, dy, dt, n_fields);
    CudaCheckError();


    // Step 4: Calculate the fluxes again
    #ifdef EXACT
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_Exact_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    #endif
    #ifdef ROE
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_Roe_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    #endif
    #ifdef HLLC
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Lx, Q_Rx, F_x, nx_s, ny_s, nz_s, n_ghost, gama, 0, n_fields);
    Calculate_HLLC_Fluxes_CUDA<<<dim2dGrid,dim1dBlock>>>(Q_Ly, Q_Ry, F_y, nx_s, ny_s, nz_s, n_ghost, gama, 1, n_fields);
    #endif
    CudaCheckError();

#endif //CTU

    #ifdef DE
    // Compute the divergence of Vel before updating the conserved array, this solves syncronization issues when adding this term on Update_Conserved_Variables
    Partial_Update_Advected_Internal_Energy_2D<<<dim2dGrid,dim1dBlock>>>( dev_conserved, Q_Lx, Q_Rx, Q_Ly, Q_Ry, nx_s, ny_s, n_ghost, dx, dy, dt, gama, n_fields );
    #endif


    // Step 5: Update the conserved variable array
    Update_Conserved_Variables_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, F_x, F_y, nx_s, ny_s, x_off_s, y_off_s, n_ghost, dx, dy, xbound, ybound, dt, gama, n_fields);
    CudaCheckError();

    // Synchronize the total and internal energy
    #ifdef DE
    Select_Internal_Energy_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, n_ghost, n_fields);    
    Sync_Energies_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, n_ghost, gama, n_fields);
    CudaCheckError();    
    #endif

    // Apply cooling
    #ifdef COOLING_GPU
    cooling_kernel<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, nz_s, n_ghost, n_fields, dt, gama, dev_dt_array);
    CudaCheckError();    
    #endif

    // Step 6: Calculate the next timestep
    Calc_dt_2D<<<dim2dGrid,dim1dBlock>>>(dev_conserved, nx_s, ny_s, n_ghost, dx, dy, dev_dti_array, gama);
    CudaCheckError();    


    // copy the conserved variable array back to the CPU
    CudaSafeCall( cudaMemcpy(tmp2, dev_conserved, n_fields*BLOCK_VOL*sizeof(Real), cudaMemcpyDeviceToHost) );

    // copy the updated conserved variable array back into the host_conserved array on the CPU
    host_return_block_2D(nx, ny, nx_s, ny_s, n_ghost, block, block1_tot, block2_tot, remainder1, remainder2, BLOCK_VOL, host_conserved1, buffer, n_fields);


    // copy the dti array onto the CPU
    CudaSafeCall( cudaMemcpy(host_dti_array, dev_dti_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // iterate through to find the maximum inverse dt for this subgrid block
    for (int i=0; i<ngrid; i++) {
      max_dti = fmax(max_dti, host_dti_array[i]);
    }
    #ifdef COOLING_GPU
    // copy the dt array from cooling onto the CPU
    CudaSafeCall( cudaMemcpy(host_dt_array, dev_dt_array, ngrid*sizeof(Real), cudaMemcpyDeviceToHost) );
    // iterate through to find the minimum dt for this subgrid block
    for (int i=0; i<ngrid; i++) {
      min_dt = fmin(min_dt, host_dt_array[i]);
    }  
    //printf("%f %f\n", min_dt, 0.3/max_dti); 
    if (min_dt < 0.3/max_dti) {
      //printf("%f %f\n", min_dt, 0.3/max_dti); 
      min_dt = fmax(min_dt, 1.0);
      max_dti = 0.3/min_dt;
    }
    #endif


    // add one to the counter
    block++;

  }


  #ifdef DYNAMIC_GPU_ALLOC
  // If memory is not single allocated then free the memory every timestep.
  Free_Memory_CTU_2D();
  #endif

  // return the maximum inverse timestep
  return max_dti;

}

void Free_Memory_CTU_2D() {

  // free the CPU memory
  if (block_tot > 1) free(buffer);
  free(host_dti_array);
  #ifdef COOLING_GPU
  free(host_dt_array);  
  #endif    

  // free the GPU memory
  cudaFree(dev_conserved);
  cudaFree(Q_Lx);
  cudaFree(Q_Rx);
  cudaFree(Q_Ly);
  cudaFree(Q_Ry);
  cudaFree(F_x);
  cudaFree(F_y);
  cudaFree(dev_dti_array);
  #ifdef COOLING_GPU
  cudaFree(dev_dt_array);
  #endif

}


__global__ void Evolve_Interface_States_2D(Real *dev_Q_Lx, Real *dev_Q_Rx, Real *dev_F_x, 
                                           Real *dev_Q_Ly, Real *dev_Q_Ry, Real *dev_F_y,
                                           int nx, int ny, int n_ghost, Real dx, Real dy, Real dt, int n_fields)
{
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  int n_cells = nx*ny;

  // get a thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  int tid = threadIdx.x + blockId * blockDim.x;
  int yid = tid / nx;
  int xid = tid - yid*nx;
  int id = xid + yid*nx;


  // set the new x interface states
  if (xid > n_ghost-2 && xid < nx-n_ghost && yid > n_ghost-2 && yid < ny-n_ghost+1)
  {
    // left
    int ipo = xid+1 + yid*nx;
    int jmo = xid + (yid-1)*nx;
    int ipojmo = xid+1 + (yid-1)*nx;
    dev_Q_Lx[            id] += 0.5*dtody*(dev_F_y[            jmo] - dev_F_y[            id]);
    dev_Q_Lx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_Q_Lx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]);
    dev_Q_Lx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_Q_Lx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Lx[(5+i)*n_cells + id] += 0.5*dtody*(dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_Q_Lx[(n_fields-1)*n_cells + id] += 0.5*dtody*(dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id]);
    #endif
    // right
    dev_Q_Rx[            id] += 0.5*dtody*(dev_F_y[            ipojmo] - dev_F_y[            ipo]);
    dev_Q_Rx[  n_cells + id] += 0.5*dtody*(dev_F_y[  n_cells + ipojmo] - dev_F_y[  n_cells + ipo]);
    dev_Q_Rx[2*n_cells + id] += 0.5*dtody*(dev_F_y[2*n_cells + ipojmo] - dev_F_y[2*n_cells + ipo]);
    dev_Q_Rx[3*n_cells + id] += 0.5*dtody*(dev_F_y[3*n_cells + ipojmo] - dev_F_y[3*n_cells + ipo]);
    dev_Q_Rx[4*n_cells + id] += 0.5*dtody*(dev_F_y[4*n_cells + ipojmo] - dev_F_y[4*n_cells + ipo]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Rx[(5+i)*n_cells + id] += 0.5*dtody*(dev_F_y[(5+i)*n_cells + ipojmo] - dev_F_y[(5+i)*n_cells + ipo]);
    }
    #endif
    #ifdef DE
    dev_Q_Rx[(n_fields-1)*n_cells + id] += 0.5*dtody*(dev_F_y[(n_fields-1)*n_cells + ipojmo] - dev_F_y[(n_fields-1)*n_cells + ipo]);
    #endif
  }
  // set the new y interface states
  if (yid > n_ghost-2 && yid < ny-n_ghost && xid > n_ghost-2 && xid < nx-n_ghost+1)
  {
    // left
    int jpo = xid + (yid+1)*nx;
    int imo = xid-1 + yid*nx;
    int jpoimo = xid-1 + (yid+1)*nx;
    dev_Q_Ly[            id] += 0.5*dtodx*(dev_F_x[            imo] - dev_F_x[            id]); 
    dev_Q_Ly[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]); 
    dev_Q_Ly[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]); 
    dev_Q_Ly[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id]); 
    dev_Q_Ly[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id]); 
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Ly[(5+i)*n_cells + id] += 0.5*dtodx*(dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id]); 
    }
    #endif
    #ifdef DE
    dev_Q_Ly[(n_fields-1)*n_cells + id] += 0.5*dtodx*(dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id]); 
    #endif
    // right
    dev_Q_Ry[            id] += 0.5*dtodx*(dev_F_x[            jpoimo] - dev_F_x[            jpo]); 
    dev_Q_Ry[  n_cells + id] += 0.5*dtodx*(dev_F_x[  n_cells + jpoimo] - dev_F_x[  n_cells + jpo]); 
    dev_Q_Ry[2*n_cells + id] += 0.5*dtodx*(dev_F_x[2*n_cells + jpoimo] - dev_F_x[2*n_cells + jpo]); 
    dev_Q_Ry[3*n_cells + id] += 0.5*dtodx*(dev_F_x[3*n_cells + jpoimo] - dev_F_x[3*n_cells + jpo]); 
    dev_Q_Ry[4*n_cells + id] += 0.5*dtodx*(dev_F_x[4*n_cells + jpoimo] - dev_F_x[4*n_cells + jpo]); 
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_Q_Ry[(5+i)*n_cells + id] += 0.5*dtodx*(dev_F_x[(5+i)*n_cells + jpoimo] - dev_F_x[(5+i)*n_cells + jpo]); 
    }
    #endif
    #ifdef DE
    dev_Q_Ry[(n_fields-1)*n_cells + id] += 0.5*dtodx*(dev_F_x[(n_fields-1)*n_cells + jpoimo] - dev_F_x[(n_fields-1)*n_cells + jpo]); 
    #endif
  }

}


#endif //CUDA

