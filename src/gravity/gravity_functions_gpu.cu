
#include <hip/hip_runtime.h>
#if defined(GRAVITY) && defined(GRAVITY_GPU)

#include"../grid3D.h"
#include"../global.h"
#include "../io.h"
#include "../error_handling.h"
#include <cstring>


void Grav3D::AllocateMemory_GPU(){
  
  CudaSafeCall( cudaMalloc((void**)&F.density_d,  n_cells*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.potential_d,   n_cells_potential*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.potential_1_d, n_cells_potential*sizeof(Real)) );
  
  #if defined(MPI_CHOLLA) && !defined(MPI_GPU)
  //Device buffers for potential transfers when the MPI_GPU is disabled
  int nGHST, nx_g, ny_g, nz_g;
  nGHST = N_GHOST_POTENTIAL;
  nx_g = nx_local + 2*nGHST;
  ny_g = ny_local + 2*nGHST;
  nz_g = nz_local + 2*nGHST;
  
  int buffer_size_x, buffer_size_y, buffer_size_z;
  buffer_size_x = nGHST * nz_g * ny_g;
  buffer_size_y = nGHST * nz_g * nx_g;
  buffer_size_z = nGHST * nx_g * ny_g; 
  
  CudaSafeCall( cudaMalloc((void**)&F.send_buffer_potential_x0_d, buffer_size_x*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.send_buffer_potential_x1_d, buffer_size_x*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.send_buffer_potential_y0_d, buffer_size_y*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.send_buffer_potential_y1_d, buffer_size_y*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.send_buffer_potential_z0_d, buffer_size_z*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.send_buffer_potential_z1_d, buffer_size_z*sizeof(Real)) );

  CudaSafeCall( cudaMalloc((void**)&F.recv_buffer_potential_x0_d, buffer_size_x*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.recv_buffer_potential_x1_d, buffer_size_x*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.recv_buffer_potential_y0_d, buffer_size_y*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.recv_buffer_potential_y1_d, buffer_size_y*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.recv_buffer_potential_z0_d, buffer_size_z*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.recv_buffer_potential_z1_d, buffer_size_z*sizeof(Real)) );
  chprintf( "Allocated Gravity GPU MPI Buffers  \n" );
  #endif//MPI_CHOLLA-MPI_GPU    
  
  #ifdef GRAVITY_GPU

  #ifdef GRAV_ISOLATED_BOUNDARY_X
  CudaSafeCall( cudaMalloc((void**)&F.pot_boundary_x0_d, N_GHOST_POTENTIAL*ny_local*nz_local*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.pot_boundary_x1_d, N_GHOST_POTENTIAL*ny_local*nz_local*sizeof(Real)) );
  #endif
  #ifdef GRAV_ISOLATED_BOUNDARY_Y
  CudaSafeCall( cudaMalloc((void**)&F.pot_boundary_y0_d, N_GHOST_POTENTIAL*nx_local*nz_local*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.pot_boundary_y1_d, N_GHOST_POTENTIAL*nx_local*nz_local*sizeof(Real)) );
  #endif
  #ifdef GRAV_ISOLATED_BOUNDARY_Z
  CudaSafeCall( cudaMalloc((void**)&F.pot_boundary_z0_d, N_GHOST_POTENTIAL*nx_local*ny_local*sizeof(Real)) );
  CudaSafeCall( cudaMalloc((void**)&F.pot_boundary_z1_d, N_GHOST_POTENTIAL*nx_local*ny_local*sizeof(Real)) );
  #endif
  
  #endif//GRAVITY_GPU
  
  chprintf( "Allocated Gravity GPU memory \n" );
}


void Grav3D::FreeMemory_GPU(void){
  
  cudaFree( F.density_d );
  cudaFree( F.potential_d );
  cudaFree( F.potential_1_d );
  
  #if defined(MPI_CHOLLA) && !defined(MPI_GPU)  
  cudaFree( F.send_buffer_potential_x0_d );
  cudaFree( F.send_buffer_potential_x1_d );
  cudaFree( F.send_buffer_potential_y0_d );
  cudaFree( F.send_buffer_potential_y1_d );
  cudaFree( F.send_buffer_potential_z0_d );
  cudaFree( F.send_buffer_potential_z1_d );
  
  cudaFree( F.recv_buffer_potential_x0_d );
  cudaFree( F.recv_buffer_potential_x1_d );
  cudaFree( F.recv_buffer_potential_y0_d );
  cudaFree( F.recv_buffer_potential_y1_d );
  cudaFree( F.recv_buffer_potential_z0_d );
  cudaFree( F.recv_buffer_potential_z1_d );
  #endif//MPI_CHOLLA-MPI_GPU   
  
  #ifdef GRAVITY_GPU
  
  #ifdef GRAV_ISOLATED_BOUNDARY_X
  cudaFree( F.pot_boundary_x0_d);
  cudaFree( F.pot_boundary_x1_d);
  #endif
  #ifdef GRAV_ISOLATED_BOUNDARY_Y
  cudaFree( F.pot_boundary_y0_d);
  cudaFree( F.pot_boundary_y1_d);
  #endif
  #ifdef GRAV_ISOLATED_BOUNDARY_Z
  cudaFree( F.pot_boundary_z0_d);
  cudaFree( F.pot_boundary_z1_d);
  #endif
  
  #endif //GRAVITY_GPU
  
}

void __global__ Copy_Hydro_Density_to_Gravity_Kernel( Real *src_density_d, Real *dst_density_d, int nx_local, int ny_local, int nz_local, int n_ghost, Real cosmo_rho_0_gas   ){
  
  int tid_x, tid_y, tid_z, tid_grid, tid_dens;
  tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  tid_y = blockIdx.y * blockDim.y + threadIdx.y;
  tid_z = blockIdx.z * blockDim.z + threadIdx.z;
  
  if (tid_x >= nx_local || tid_y >= ny_local || tid_z >= nz_local ) return;  
  
  tid_dens = tid_x + tid_y*nx_local + tid_z*nx_local*ny_local;
  
  tid_x += n_ghost;
  tid_y += n_ghost;
  tid_z += n_ghost;

  int nx_grid, ny_grid;
  nx_grid = nx_local + 2*n_ghost;
  ny_grid = ny_local + 2*n_ghost;
  tid_grid = tid_x + tid_y*nx_grid + tid_z*nx_grid*ny_grid;
  
  Real dens;
  dens = src_density_d[tid_grid];
  
  #ifdef COSMOLOGY
  dens *= cosmo_rho_0_gas;
  #endif
  
  #ifdef PARTICLES
  dst_density_d[tid_dens] += dens; //Hydro density is added AFTER partices density  
  #else
  dst_density_d[tid_dens]  = dens;
  #endif
             
}

void Grid3D::Copy_Hydro_Density_to_Gravity_GPU(){
  
  int nx_local, ny_local, nz_local, n_ghost;
  nx_local = Grav.nx_local;
  ny_local = Grav.ny_local;
  nz_local = Grav.nz_local;
  n_ghost  = H.n_ghost;
  
  
  
  // set values for GPU kernels
  int tpb_x = TPBX_GRAV;
  int tpb_y = TPBY_GRAV;
  int tpb_z = TPBZ_GRAV;
  int ngrid_x = (nx_local - 1) / tpb_x + 1;
  int ngrid_y = (ny_local - 1) / tpb_y + 1;
  int ngrid_z = (nz_local - 1) / tpb_z + 1;
  // number of blocks per 1D grid  
  dim3 dim3dGrid(ngrid_x, ngrid_y, ngrid_z);
  //  number of threads per 1D block   
  dim3 dim3dBlock(tpb_x, tpb_y, tpb_z);
  
  Real cosmo_rho_0_gas;
  
  #ifdef COSMOLOGY
  cosmo_rho_0_gas = Cosmo.rho_0_gas;
  #else
  cosmo_rho_0_gas = 1.0;
  #endif
  
  #ifndef MPI_GPU
  //Copy the hydro density from host to device
  int n_cells_total = ( nx_local + 2*n_ghost ) * ( ny_local + 2*n_ghost ) * ( nz_local + 2*n_ghost );
  CudaSafeCall( cudaMemcpy(C.d_density, C.density, n_cells_total*sizeof(Real), cudaMemcpyHostToDevice) ); 
  #endif//MPI_GPU
   
  //Copy the density from the device array to the Poisson input density array
  hipLaunchKernelGGL(Copy_Hydro_Density_to_Gravity_Kernel, dim3dGrid, dim3dBlock, 0, 0,  C.d_density, Grav.F.density_d, nx_local, ny_local, nz_local, n_ghost, cosmo_rho_0_gas);
  
  
}

void __global__ Extrapolate_Grav_Potential_Kernel( Real *dst_potential, Real *src_potential_0, Real *src_potential_1, 
        int nx_pot, int ny_pot, int nz_pot, int nx_grid, int ny_grid, int nz_grid, int n_offset, 
        Real dt_now, Real dt_prev, bool INITIAL,  Real cosmo_factor ){
                      
  int tid_x, tid_y, tid_z, tid_grid, tid_pot;
  tid_x = blockIdx.x * blockDim.x + threadIdx.x;
  tid_y = blockIdx.y * blockDim.y + threadIdx.y;
  tid_z = blockIdx.z * blockDim.z + threadIdx.z;

  if (tid_x >= nx_pot || tid_y >= ny_pot || tid_z >= nz_pot ) return;  

  tid_pot = tid_x + tid_y*nx_pot + tid_z*nx_pot*ny_pot;                
  
  tid_x += n_offset;
  tid_y += n_offset;
  tid_z += n_offset;

  tid_grid = tid_x + tid_y*nx_grid + tid_z*nx_grid*ny_grid;
  
  Real pot_now, pot_prev, pot_extrp;
  pot_now = src_potential_0[tid_pot]; //Potential at the n-th timestep
  if ( INITIAL ){
    pot_extrp = pot_now; //The first timestep the extrapolated potential is phi_0
  } else {
    pot_prev = src_potential_1[tid_pot]; //Potential at the (n-1)-th timestep ( previous step )
    //Compute the extrapolated potential from phi_n-1 and phi_n
    pot_extrp = pot_now  + 0.5 * dt_now * ( pot_now - pot_prev  ) / dt_prev;
  }
  
  #ifdef COSMOLOGY
  //For cosmological simulation the potential is transformrd to 'comuving coordinates' 
  pot_extrp *= cosmo_factor;
  #endif
  
  //Save the extrapolated potential
  dst_potential[tid_grid] = pot_extrp;
  //Set phi_n-1 = phi_n, to use it during the next step 
  src_potential_1[tid_pot] = pot_now;
}

void Grid3D::Extrapolate_Grav_Potential_GPU(){
  
  int nx_pot, ny_pot, nz_pot;
  nx_pot = Grav.nx_local + 2*N_GHOST_POTENTIAL;
  ny_pot = Grav.ny_local + 2*N_GHOST_POTENTIAL;
  nz_pot = Grav.nz_local + 2*N_GHOST_POTENTIAL;
  
  int n_ghost_grid, nx_grid, ny_grid, nz_grid;
  n_ghost_grid = H.n_ghost;
  nx_grid = Grav.nx_local + 2*n_ghost_grid;
  ny_grid = Grav.ny_local + 2*n_ghost_grid;
  nz_grid = Grav.nz_local + 2*n_ghost_grid;
  
  int n_offset = n_ghost_grid - N_GHOST_POTENTIAL;
  
  
  Real dt_now, dt_prev, cosmo_factor;
  dt_now = Grav.dt_now;
  dt_prev = Grav.dt_prev;
  
  #ifdef COSMOLOGY
  cosmo_factor = Cosmo.current_a * Cosmo.current_a / Cosmo.phi_0_gas;
  #else
  cosmo_factor = 1.0;
  #endif  
  
  // set values for GPU kernels
  int tpb_x = TPBX_GRAV;
  int tpb_y = TPBY_GRAV;
  int tpb_z = TPBZ_GRAV;
  int ngrid_x = (nx_pot - 1) / tpb_x + 1;
  int ngrid_y = (ny_pot - 1) / tpb_y + 1;
  int ngrid_z = (nz_pot - 1) / tpb_z + 1;
  // number of blocks per 1D grid  
  dim3 dim3dGrid(ngrid_x, ngrid_y, ngrid_z);
  //  number of threads per 1D block   
  dim3 dim3dBlock(tpb_x, tpb_y, tpb_z);
  
  hipLaunchKernelGGL(Extrapolate_Grav_Potential_Kernel, dim3dGrid, dim3dBlock, 0, 0, C.d_Grav_potential, Grav.F.potential_d, Grav.F.potential_1_d, nx_pot, ny_pot, nz_pot, nx_grid, ny_grid, nz_grid, n_offset, dt_now, dt_prev, Grav.INITIAL, cosmo_factor );

}





#endif //GRAVITY
