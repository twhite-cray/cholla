
#include <hip/hip_runtime.h>
#if defined(PARTICLES) && defined(PARTICLES_GPU)

#include <unistd.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include"gpu.hpp"
#include"../io.h"
#include"../global.h"
#include"../global_cuda.h"
#include "particles_3D.h"



void Particles_3D::Free_GPU_Array_Real( Real *array ){ cudaFree(array); }
void Particles_3D::Free_GPU_Array_int( int *array )  { cudaFree(array); }
void Particles_3D::Free_GPU_Array_bool( bool *array ){ cudaFree(array); }


void __global__ Copy_Device_to_Device_Kernel( Real *src_array_dev, Real *dst_array_dev, part_int_t size ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x ;
  if ( tid < size ) dst_array_dev[tid] = src_array_dev[tid];  
}

void Copy_Device_to_Device( Real *src_array_dev, Real *dst_array_dev, part_int_t size ){
  int ngrid =  (size + TPB_PARTICLES - 1) / TPB_PARTICLES;
  dim3 dim1dGrid(ngrid, 1, 1);
  dim3 dim1dBlock(TPB_PARTICLES, 1, 1);
  hipLaunchKernelGGL(Copy_Device_to_Device_Kernel, dim1dGrid, dim1dBlock, 0, 0,  src_array_dev, dst_array_dev, size);
  CudaCheckError();
  
}

void Particles_3D::Reallocate_and_Copy_Partciles_Array_Real( Real **src_array_dev, part_int_t size_initial, part_int_t size_end  ){
  size_t global_free, global_total;
  CudaSafeCall( cudaMemGetInfo( &global_free, &global_total ) );
  cudaDeviceSynchronize();
  #ifdef PRINT_GPU_MEMORY
  printf( "ReAlocating GPU Memory:  %ld  MB free \n", global_free/1000000);
  #endif
  if ( global_free < size_end*sizeof(Real) ){
    printf( "ERROR: Not enough global device memory \n" );
    printf( " Available Memory: %ld  MB \n", global_free/1000000  );
    printf( " Requested Memory: %ld  MB \n", size_end*sizeof(Real)/1000000  );
    exit(-1);
  }
  Real *temp_array_dev;
  CudaSafeCall( cudaMalloc((void**)&temp_array_dev,  size_end*sizeof(Real)) );
  cudaDeviceSynchronize();
  // printf( " Alocated GPU Memory:  %ld  MB \n", size_end*sizeof(Real)/1000000 );
  if ( size_initial*sizeof(Real) > size_end*sizeof(Real) ){
    printf("ERROR: Memory to copy larger than array size\n" );
    exit(-1);
  }
  // printf( " Copying:  %ld  ->  %ld  \n", size_initial*sizeof(Real), size_end*sizeof(Real) );
  // CudaSafeCall( cudaMemcpy(temp_array_dev, *src_array_dev, size_initial*sizeof(Real), cudaMemcpyDeviceToDevice) );
  // NOTE: cudaMemcpy is not working! made kernel to do the device to device copy
  Copy_Device_to_Device( *src_array_dev, temp_array_dev,  size_initial );
  cudaDeviceSynchronize();
  CudaSafeCall( cudaFree( *src_array_dev ));
  cudaDeviceSynchronize();
  *src_array_dev = temp_array_dev;
  
}




void Particles_3D::Allocate_Particles_GPU_Array_Real( Real **array_dev, part_int_t size ){
  size_t global_free, global_total;
  CudaSafeCall( cudaMemGetInfo( &global_free, &global_total ) );
  #ifdef PRINT_GPU_MEMORY
  chprintf( "Allocating GPU Memory:  %ld  MB free \n", global_free/1000000);
  #endif
  if ( global_free < size*sizeof(Real) ){
    printf( "ERROR: Not enough global device memory \n" );
    printf( " Available Memory: %ld  MB \n", global_free/1000000  );
    printf( " Requested Memory: %ld  MB \n", size*sizeof(Real)/1000000  );
    exit(-1);
  }
  CudaSafeCall( cudaMalloc((void**)array_dev,  size*sizeof(Real)) );
  cudaDeviceSynchronize();
}

void Particles_3D::Allocate_Particles_Grid_Field_Real( Real **array_dev, int size ){
  size_t global_free, global_total;
  CudaSafeCall( cudaMemGetInfo( &global_free, &global_total ) );
  #ifdef PRINT_GPU_MEMORY
  chprintf( "Allocating GPU Memory:  %ld  MB free \n", global_free/1000000);
  #endif
  if ( global_free < size*sizeof(Real) ){
    printf( "ERROR: Not enough global device memory \n" );
    printf( " Available Memory: %ld  MB \n", global_free/1000000  );
    printf( " Requested Memory: %ld  MB \n", size*sizeof(Real)/1000000  );
    exit(-1);
  }
  CudaSafeCall( cudaMalloc((void**)array_dev,  size*sizeof(Real)) );
  cudaDeviceSynchronize();
}

void Particles_3D::Allocate_Particles_GPU_Array_int( int **array_dev, part_int_t size ){
  size_t global_free, global_total;
  CudaSafeCall( cudaMemGetInfo( &global_free, &global_total ) );
  #ifdef PRINT_GPU_MEMORY
  chprintf( "Allocating GPU Memory:  %ld  MB free \n", global_free/1000000);
  #endif
  if ( global_free < size*sizeof(int) ){
    printf( "ERROR: Not enough global device memory \n" );
    printf( " Available Memory: %ld  MB \n", global_free/1000000  );
    printf( " Requested Memory: %ld  MB \n", size*sizeof(int)/1000000  );
    exit(-1);
  }
  CudaSafeCall( cudaMalloc((void**)array_dev,  size*sizeof(int)) );
  cudaDeviceSynchronize();
}

void Particles_3D::Allocate_Particles_GPU_Array_bool( bool **array_dev, part_int_t size ){
  size_t global_free, global_total;
  CudaSafeCall( cudaMemGetInfo( &global_free, &global_total ) );
  #ifdef PRINT_GPU_MEMORY
  chprintf( "Allocating GPU Memory:  %ld  MB free \n", global_free/1000000);
  #endif
  if ( global_free < size*sizeof(bool) ){
    printf( "ERROR: Not enough global device memory \n" );
    printf( " Available Memory: %ld  MB \n", global_free/1000000  );
    printf( " Requested Memory: %ld  MB \n", size*sizeof(bool)/1000000  );
    exit(-1);
  }
  CudaSafeCall( cudaMalloc((void**)array_dev,  size*sizeof(bool)) );
  cudaDeviceSynchronize();
}

void Particles_3D::Copy_Particles_Array_Real_Host_to_Device( Real *array_host, Real *array_dev, part_int_t size){
  CudaSafeCall( cudaMemcpy(array_dev, array_host, size*sizeof(Real), cudaMemcpyHostToDevice) );
  cudaDeviceSynchronize();
}

void Particles_3D::Copy_Particles_Array_Real_Device_to_Host( Real *array_dev, Real *array_host, part_int_t size){
  CudaSafeCall( cudaMemcpy(array_host, array_dev, size*sizeof(Real), cudaMemcpyDeviceToHost) );
  cudaDeviceSynchronize();
}



__global__ void Set_Particles_Array_Real_Kernel( Real value, Real *array_dev, part_int_t size ){
  int tid = blockIdx.x * blockDim.x + threadIdx.x ;
  if ( tid < size ) array_dev[tid] = value;
}



void Particles_3D::Set_Particles_Array_Real( Real value, Real *array_dev, part_int_t size){
  
  // set values for GPU kernels
  int ngrid =  (size + TPB_PARTICLES - 1) / TPB_PARTICLES;
  // number of blocks per 1D grid  
  dim3 dim1dGrid(ngrid, 1, 1);
  //  number of threads per 1D block   
  dim3 dim1dBlock(TPB_PARTICLES, 1, 1);
  hipLaunchKernelGGL(Set_Particles_Array_Real_Kernel, dim1dGrid, dim1dBlock, 0, 0,  value, array_dev, size);
  CudaCheckError();
}








#endif//PARTICLES